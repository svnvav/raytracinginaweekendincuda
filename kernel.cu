#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "vec3.h"
#include ""

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__global__ void render(vec3 *fb, int max_x, int max_y) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	fb[pixel_index] = vec3(float(i) / max_x, float(j) / max_y, 0.2);
}

int main()
{
	int nx = 1024;
	int ny = 768;

	int tx = 8;
	int ty = 8;

	int num_pixels = nx * ny;
	size_t fb_size = num_pixels * sizeof(vec3);

	vec3 *fb;
	checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render <<<blocks, threads>>> (fb, nx, ny);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	std::ofstream outfile;
	outfile.open("HelloWorld.ppm");
	outfile << "P3\n" << nx << " " << ny << "\n255\n";

	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			size_t pixel_index = j * nx + i;
			int ir = int(255.99 * fb[pixel_index][0]);
			int ig = int(255.99 * fb[pixel_index][1]);
			int ib = int(255.99 * fb[pixel_index][2]);
			outfile << ir << " " << ig << " " << ib << "\n";
		}
	}
	checkCudaErrors(hipFree(fb));

    return 0;
}


